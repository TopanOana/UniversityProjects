#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "ImageProcessingFunctions.h"
#include <opencv2/core/base.hpp>
#include <opencv2/core/mat.hpp>


#include <stdio.h>
#include <vector>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <chrono>
#include <iostream>

#define BIN_WIDTH 1
#define NUMBINS 180 / BIN_WIDTH
#define KERNEL_SIZE 3
#define THRESHOLD 50
#define RATIO 3


void polarToCartesian(double rho, int theta, cv::Point& p1, cv::Point& p2) {

	int x0 = cvRound(rho * cos(theta));
	int y0 = cvRound(rho * sin(theta));

	p1.x = cvRound(x0 + 1000 * (-sin(theta)));
	p1.y = cvRound(y0 + 1000 * (cos(theta)));

	p2.x = cvRound(x0 - 1000 * (-sin(theta)));
	p2.y = cvRound(y0 - 1000 * (cos(theta)));
}


__global__ void houghTKernel(uchar* imageWEdges, int rows, int cols, int* votes, int chunkHeight, int maxChunkHeight, int remainingRows) {

	int initialRow = threadIdx.x * chunkHeight;
	int currentChunkHeight = (threadIdx.x < remainingRows) ? maxChunkHeight : chunkHeight;
	int lastRow = initialRow + currentChunkHeight;

	int currentMaxDistance = sqrtf((lastRow - initialRow + 1) * (lastRow - initialRow + 1) + cols * cols);


	for (int i = initialRow; i < lastRow; i++) {
		for (int j = 0; j < cols; j++) {
			if (imageWEdges[i * cols + j] == 255) {
				for (int theta = 0; theta < 180; theta += BIN_WIDTH) {
					int rho = round(j * cosf(theta - 90) + i * sinf(theta - 90)) + currentMaxDistance;

					votes[threadIdx.x * rho * NUMBINS + theta]++;
				}
			}
		}
	}

}


hipError_t cudaHoughTransform(cv::Mat sourceImage, int nrChunks) {

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipError_t cudaStatus;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}

	cv::Mat imageWEdges;
	Canny(sourceImage, imageWEdges, 50, 200, 3, false);


	int chunkHeight = imageWEdges.rows / nrChunks;
	int remainingRows = imageWEdges.rows % nrChunks;

	int maxChunkHeight = chunkHeight + (remainingRows > 0 ? 1 : 0);

	//allocate space for votes

	//nrchunks * sqrt(nrRowsInChunk*nrRowsInChunk + cols*cols) * NUMBINS

	int allocatedSpace = nrChunks * 2 * sqrt(maxChunkHeight * maxChunkHeight + imageWEdges.cols * imageWEdges.cols) * NUMBINS;

	//int maxDistance = sqrt(imageWEdges.rows * imageWEdges.rows + imageWEdges.cols * imageWEdges.cols);

	// Allocate and initialize votesHost
	int* votesHost = (int*)malloc(sizeof(int) * allocatedSpace);

	for (int i = 0; i < allocatedSpace; i++) {
		votesHost[i] = 0;
	}


	// Allocate votesDevice as a 2D array
	int* votesDevice;
	cudaStatus = hipMalloc(&votesDevice, sizeof(int) * allocatedSpace);

	cudaStatus = hipMemcpy(votesDevice, votesHost, sizeof(int) * allocatedSpace, hipMemcpyHostToDevice);



	//allocate space for image

	uchar* imageWEgesDevice;
	cudaStatus = hipMalloc(&imageWEgesDevice, imageWEdges.rows * imageWEdges.cols * sizeof(uchar));


	cudaStatus = hipMemcpy(imageWEgesDevice, imageWEdges.data, imageWEdges.rows * imageWEdges.cols, hipMemcpyHostToDevice);

	hipEventRecord(start);


	//launching kernels
	houghTKernel << <1, nrChunks >> > (imageWEgesDevice, imageWEdges.rows, imageWEdges.cols, votesDevice, chunkHeight, maxChunkHeight, remainingRows);

	hipDeviceSynchronize();

	hipEventRecord(stop);
	hipEventSynchronize(stop);


	cudaStatus = hipMemcpy(votesHost, votesDevice, allocatedSpace, hipMemcpyDeviceToHost);

	std::vector<cv::Mat> finalChunks;
	int value = 2 * sqrt(maxChunkHeight * maxChunkHeight + imageWEdges.cols * imageWEdges.cols);
	int lineThreshold = 150;

	for (int i = 0; i < nrChunks; i++) {
		int currentChunkHeight = (i < remainingRows) ? maxChunkHeight : chunkHeight;

		cv::Rect rect(0, i * chunkHeight, imageWEdges.cols, currentChunkHeight);
		cv::Mat chunk(imageWEdges, rect);
		if (currentChunkHeight < maxChunkHeight) {
			// Resize the chunk to have the same number of rows as maxChunkHeight
			cv::Mat resizedChunk;
			resize(chunk, resizedChunk, cv::Size(imageWEdges.cols, maxChunkHeight));
			chunk = resizedChunk;
		}
		cv::Mat finol;
		cv::cvtColor(chunk, finol, cv::ColorConversionCodes::COLOR_GRAY2BGR);


		///actual check for lines
		for (int x = 0; x < value; x++) {
			for (int y = 0; y < NUMBINS; y++) {
				if (votesHost[nrChunks * value + y] >= lineThreshold) {
					int rho = x - value / 2;
					int theta = y - 90;

					cv::Point p1, p2;
					polarToCartesian(rho, theta, p1, p2);

					line(finol, p1, p2, cv::Scalar(0, 0, 255), 2, cv::LineTypes::LINE_AA);
				}
			}
		}

		finalChunks.push_back(finol);
	}


	cv::Mat linesImage = concatenateChunksVertically(finalChunks);

	imshow("source image", sourceImage);
	imshow("detected lines", linesImage);

	// Calculate and print the elapsed time
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout << "cuda Time taken: " << milliseconds << " ms" << std::endl;



	cv::waitKey(0);

Error:

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(imageWEgesDevice);

	free(votesHost);
	hipFree(votesDevice);

	return cudaStatus;
}



int main()
{
	std::string filename("C:\\Users\\Oana\\source\\repos\\ProjectPDP\\image.png");

	cv::Mat sourceImage;

	sourceImage = cv::imread(filename, cv::ImreadModes::IMREAD_GRAYSCALE);
	
	

	// Add vectors in parallel.
	hipError_t cudaStatus = cudaHoughTransform(sourceImage, 9);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "houghTransformWithCuda failed!");
		return 1;
	}


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

